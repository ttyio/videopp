#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "cudaProcessFrame.h"

#define MUL(x,y)    (x*y)

__device__ void YUV2RGB(uint32 *yuvi, float *rgb)
{
    float luma, chromaCb, chromaCr;

    luma     = (float)yuvi[0];
    chromaCb = (float)((int32)yuvi[1] - 512.0f);
    chromaCr = (float)((int32)yuvi[2] - 512.0f);

    
    rgb[0] = MUL(luma,     1.1644) +
            MUL(chromaCb, 0) +
            MUL(chromaCr, 1.5960);
    rgb[1] = MUL(luma,     1.1644) +
            MUL(chromaCb, -0.3918) +
            MUL(chromaCr, -0.8130);
    rgb[2] = MUL(luma,    1.1644) +
            MUL(chromaCb, 2.0172) +
            MUL(chromaCr, 0);
}

__device__ void RGB2YUV(uint32 *rgb, float *yuv)
{

    float r = (float)rgb[0];
    float g = (float)rgb[1];
    float b = (float)rgb[2];

    yuv[0]  = MUL(r, 0.2568) +
            MUL(g, 0.5041) +
            MUL(b, 0.0979) + 16;
    yuv[1]= MUL(r, -0.1482) +
            MUL(g, -0.2910) +
            MUL(b, 0.4392) + 128;
    yuv[2] = MUL(r, 0.4392) +
            MUL(g, -0.3678) +
            MUL(b, -0.0714) + 128;
}


__device__ uint32 RGBAPACK_10bit(float* rgb)
{
    uint32 ARGBpixel = 0;

    rgb[0] = min(max(rgb[0], 0.0f), 1023.f);
    rgb[1] = min(max(rgb[1], 0.0f), 1023.f);
    rgb[2] = min(max(rgb[2], 0.0f), 1023.f);

    ARGBpixel = (((uint32)rgb[2]  >> 2) |
                 (((uint32)rgb[1] >> 2) << 8)  |
                 (((uint32)rgb[0]   >> 2) << 16) | ((uint32)0xff<< 24));

    return  ARGBpixel;
}

__device__ void RGBAUNPACK_10bit(uint32 pixel, uint32* rgb)
{
    rgb[2] = (pixel & 0xFF) << 2;
    rgb[1] = ((pixel>>8) & 0xFF) << 2;
    rgb[0] = ((pixel>>16) & 0xFF) << 2;
}

extern "C" __global__ void NV12ToARGBdrvapi(uint32 *srcImage,     size_t nSourcePitch,
                                  uint32 *dstImage,     size_t nDestPitch,
                                  uint32 width,         uint32 height)
{
    // process 2 pixels per thread
    int32 x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
    int32 y = blockIdx.y *  blockDim.y       +  threadIdx.y;
    if (x+1 >= width || y >= height)
        return; 

    uint32 processingPitch = nSourcePitch;
    uint32 dstImagePitch   = nDestPitch >> 2;
    uint8 *srcImageU8     = (uint8 *)srcImage;

    // Read 2 Y components at a time
    uint32 yuvi[6];
    yuvi[0] = (srcImageU8[y * processingPitch + x    ]) << 2;
    yuvi[3] = (srcImageU8[y * processingPitch + x + 1]) << 2;

    uint32 chromaOffset    = processingPitch * height;
    int32 y_chroma = y >> 1;

    if (y & 1)  // odd scanline 
    {
        uint32 chromaCb = srcImageU8[chromaOffset + y_chroma * processingPitch + x    ];
        uint32 chromaCr = srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1];

        if (y_chroma < ((height >> 1) - 1)) // interpolate vertically
        {
            chromaCb = (chromaCb + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x    ] + 1) >> 1;
            chromaCr = (chromaCr + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x + 1] + 1) >> 1;
        }

        yuvi[1] = yuvi[4] = chromaCb << 2;
        yuvi[2] = yuvi[5] = chromaCr << 2;
    }
    else
    {
        yuvi[1] = yuvi[4] = (uint32)srcImageU8[chromaOffset + y_chroma * processingPitch + x    ] << 2;
        yuvi[2] = yuvi[5] = (uint32)srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1] << 2;
    }

    // YUV to RGB Transformation conversion
    float rgb[6];
    YUV2RGB(&yuvi[0], &rgb[0]);
    YUV2RGB(&yuvi[3], &rgb[3]);

    // save to dest
    dstImage[y * dstImagePitch + x     ] = RGBAPACK_10bit(&rgb[0]);
    dstImage[y * dstImagePitch + x + 1 ] = RGBAPACK_10bit(&rgb[3]);
}

extern "C" __global__ void ARGBToNv12drvapi(uint32 *srcImage,     size_t nSourcePitch,
                                  uint32 *dstImage,     size_t nDestPitch,
                                  uint32 width,         uint32 height)
{
    int32 x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
    int32 y = blockIdx.y *  blockDim.y       +  threadIdx.y;
    if (x+1 >= width || y >= height)
        return; 

    uint32 processingPitch = nSourcePitch>>2;
    uint8 *dstImageU8     = (uint8 *)dstImage;

    uint32 rgb[6];
    float yuv[6];

    // Clamp the results to RGBA
    RGBAUNPACK_10bit(srcImage[y * processingPitch + x     ], &rgb[0]);
    RGBAUNPACK_10bit(srcImage[y * processingPitch + x + 1 ], &rgb[3]);

    // YUV to RGB Transformation conversion
    RGB2YUV(&rgb[0], &yuv[0]);
    RGB2YUV(&rgb[3], &yuv[3]);

    yuv[0]   = min(max(yuv[0]/4,   0.0f), 255.f);
    yuv[1]   = min(max((yuv[1])/4,   0.0f), 255.f);
    yuv[2]   = min(max((yuv[2])/4,   0.0f), 255.f);
    yuv[3]   = min(max(yuv[3]/4,   0.0f), 255.f);
    yuv[4]   = min(max((yuv[4])/4,   0.0f), 255.f);
    yuv[5]   = min(max((yuv[5])/4,   0.0f), 255.f);

    uint32 dstImagePitch   = nDestPitch;
    dstImageU8[y * dstImagePitch + x] = ((uint32)yuv[0]);
    dstImageU8[y * dstImagePitch + x + 1] = ((uint32)yuv[3]);

    if (y & 1){
    } else {
        int32 y_chroma = y >> 1;
        uint32 chromaOffset    = dstImagePitch* height;
        dstImageU8[chromaOffset + y_chroma * dstImagePitch + x] =  (uint8)((uint32)yuv[1]);
        dstImageU8[chromaOffset + y_chroma * dstImagePitch + x+1] =  (uint8)((uint32)yuv[2]);
    }
}

extern "C" __global__ void ARGBpostprocess(uint32 *srcImage, size_t pitch, uint32 width, uint32 height)
{
    int32 x = blockIdx.x *  blockDim.x + threadIdx.x;
    int32 y = blockIdx.y *  blockDim.y + threadIdx.y;
    if (x >= width || y >= height)
        return; 

    uint32 processingPitch = pitch>>2;
    uint32 rgb[3];
    RGBAUNPACK_10bit(srcImage[y*processingPitch + x], rgb);

    //todo

    srcImage[y*processingPitch + x] = RGBAPACK_10bit((float*)rgb);
}


